#include "hip/hip_runtime.h"
#define GRID_MAX_DIMENSIONS 151
#define GMD GRID_MAX_DIMENSIONS
/**
 * Converts a continuous location to a discrete grid position
 */
template<typename Mi, typename Mo>
__device__ __forceinline__ glm::ivec3 toGrid(flamegpu::DeviceAPI<Mi, Mo>* FLAMEGPU, const glm::vec3& location) {
    const flamegpu::DeviceEnvironment &env = FLAMEGPU->environment;  // Have to split this out otherwise template keyword required before getProperty
    const float R_voxel = env.getProperty<float>("R_voxel");
    const glm::uvec3 grid_dims = env.getProperty<glm::uvec3>("grid_dims");
    const glm::vec3 span = glm::vec3(grid_dims) * R_voxel * 2.0f;
    const glm::uvec3 grid_origin = env.getProperty<glm::uvec3>("grid_origin");
    return glm::ivec3(
        grid_origin.x + floor((location.x + span.x / 2.0f) / R_voxel / 2.0f),
        grid_origin.y + floor((location.y + span.y / 2.0f) / R_voxel / 2.0f),
        grid_origin.z + floor((location.z + span.z / 2.0f) / R_voxel / 2.0f));
}
/**
 * Notify the SC tracking grid counters of the passed cell's state
 */
template<typename Mi, typename Mo>
__device__ void increment_grid_sc(flamegpu::DeviceAPI<Mi, Mo>* FLAMEGPU, const glm::ivec3& gid) {
    const flamegpu::DeviceEnvironment& env = FLAMEGPU->environment;  // Have to split this out otherwise template keyword required before getProperty
    // Notify that we are present
    ++env.getMacroProperty<unsigned int, GMD, GMD, GMD>("Nsc_grid")[gid.x][gid.y][gid.z];
    if (FLAMEGPU->template getVariable<int>("apop") == 1) {
        // ++env.getMacroProperty<unsigned int, GMD, GMD, GMD>("Nsca_grid")[gid.x][gid.y][gid.z];
    } else if (FLAMEGPU->template getVariable<int>("necro") == 1) {
        ++env.getMacroProperty<unsigned int, GMD, GMD, GMD>("Nscn_grid")[gid.x][gid.y][gid.z];
    } else {
        ++env.getMacroProperty<unsigned int, GMD, GMD, GMD>("Nscl_grid")[gid.x][gid.y][gid.z];
        if (FLAMEGPU->template getVariable<int>("neighbours") < env.getProperty<int>("N_neighbours")) {
            ++env.getMacroProperty<unsigned int, GMD, GMD, GMD>("Nscl_col_grid")[gid.x][gid.y][gid.z];
        }
    }
}

FLAMEGPU_AGENT_FUNCTION(output_matrix_grid_cell, flamegpu::MessageNone, flamegpu::MessageNone) {
    const glm::ivec3 gid = toGrid(FLAMEGPU, FLAMEGPU->getVariable<glm::vec3>("xyz"));
    increment_grid_sc(FLAMEGPU, gid);
    return flamegpu::ALIVE;
}