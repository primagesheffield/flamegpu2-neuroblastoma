#include "hip/hip_runtime.h"
#include <glm/gtx/component_wise.hpp>

#include "header.h"

FLAMEGPU_CUSTOM_REDUCTION(glm_min, a, b) {
    return glm::min(a, b);
}
FLAMEGPU_CUSTOM_REDUCTION(glm_max, a, b) {
    return glm::max(a, b);
}
FLAMEGPU_HOST_FUNCTION(CAexpand) {
    // Calculate average matrix value incase the grid increases
    // Initially matrix_dummy is init correct, but grid_dims isn't, so don't calc until CAexpand() has been called once.
    static bool first_time = true;
    if (!first_time) {
        const glm::uvec3 grid_dims = FLAMEGPU->environment.getProperty<glm::uvec3>("grid_dims");
        const float matrix_dummy = FLAMEGPU->agent("GridCell").sum<float>("matrix_value") / compMul(grid_dims);
        FLAMEGPU->environment.setProperty<float>("matrix_dummy", matrix_dummy);
        first_time = false;
    }
    // Expand the grid to match tumour growth.
    // 1. Check if the tumour has expanded in each direction.
    // 2. If so, expand the matrix grid in that direction, using the average quantity in the existing grid.
    // 3. Expand the grids for various cell types.
    const glm::ivec3 oldspan = FLAMEGPU->environment.getProperty<glm::uvec3>("grid_span");
    FLAMEGPU->environment.setProperty<glm::uvec3>("grid_span_old", oldspan);

    glm::ivec3 newspan;
    {
        const glm::vec3 min_pos = min(
            FLAMEGPU->agent("Neuroblastoma").reduce<glm::vec3>("xyz", glm_min, glm::vec3(std::numeric_limits<float>().max())),
            FLAMEGPU->agent("Schwann").reduce<glm::vec3>("xyz", glm_min, glm::vec3(std::numeric_limits<float>().max())));
        const glm::vec3 max_pos = max(
            FLAMEGPU->agent("Neuroblastoma").reduce<glm::vec3>("xyz", glm_max, glm::vec3(-std::numeric_limits<float>().max())),
            FLAMEGPU->agent("Schwann").reduce<glm::vec3>("xyz", glm_max, glm::vec3(-std::numeric_limits<float>().max())));
        const float R_voxel = FLAMEGPU->environment.getProperty<float>("R_voxel");
        newspan = glm::uvec3(glm::ceil((glm::max(glm::abs(min_pos), glm::abs(max_pos)) / R_voxel / 2.0f) + 0.5f));
    }
    // clamp span (i don't think the python algorithm lets o2 grid shrink)
    const glm::uvec3 newspan_u = max(oldspan, newspan);
    FLAMEGPU->environment.setProperty<glm::uvec3>("grid_span", newspan_u);
    const glm::uvec3 new_grid_dims = (newspan_u * 2u) - glm::uvec3(1);
    FLAMEGPU->environment.setProperty<glm::uvec3>("grid_dims", new_grid_dims);
    if (new_grid_dims.x > GRID_MAX_DIMENSIONS || new_grid_dims.y > GRID_MAX_DIMENSIONS || new_grid_dims.z > GRID_MAX_DIMENSIONS) {
        fprintf(stderr, "grid has grown too large (%u, %u, %u), recompile with bigger!\n", new_grid_dims.x, new_grid_dims.y, new_grid_dims.z);
        const glm::vec3 min_pos = min(
            FLAMEGPU->agent("Neuroblastoma").reduce<glm::vec3>("xyz", glm_min, glm::vec3(std::numeric_limits<float>().max())),
            FLAMEGPU->agent("Schwann").reduce<glm::vec3>("xyz", glm_min, glm::vec3(std::numeric_limits<float>().max())));
        const glm::vec3 max_pos = max(
            FLAMEGPU->agent("Neuroblastoma").reduce<glm::vec3>("xyz", glm_max, glm::vec3(-std::numeric_limits<float>().max())),
            FLAMEGPU->agent("Schwann").reduce<glm::vec3>("xyz", glm_max, glm::vec3(-std::numeric_limits<float>().max())));
        fprintf(stderr, "step %u\n", FLAMEGPU->getStepCounter());
        fprintf(stderr, "Min Pos (%f, %f, %f), MaxPos (%f, %f, %f)!\n", min_pos.x, min_pos.y, min_pos.z, max_pos.x, max_pos.y, max_pos.z);
        throw std::runtime_error("grid has grown too large, recompile with bigger\n");
    }
    glm::uvec3 grid_origin = (glm::uvec3(GRID_MAX_DIMENSIONS) - new_grid_dims) / 2u;
    FLAMEGPU->environment.setProperty<glm::uvec3>("grid_origin", grid_origin);
}
FLAMEGPU_HOST_FUNCTION(alter2) {
    const unsigned int Nnbl_count = FLAMEGPU->agent("GridCell").sum<unsigned int>("Nnbl_grid");
    const unsigned int Nscl_count = FLAMEGPU->agent("GridCell").sum<unsigned int>("Nscl_grid");
    FLAMEGPU->environment.setProperty<unsigned int>("Nnbl_count", Nnbl_count);
    FLAMEGPU->environment.setProperty<unsigned int>("Nscl_count", Nscl_count);

    // Don't update O2/matrix if it's INIT pass
    if (FLAMEGPU->getStepCounter() == 0) {
        FLAMEGPU->environment.setProperty<int>("SKIP_ALTER", 0);
        return;
    }

    if (FLAMEGPU->environment.getProperty<int>("staticO2") == 0) {
        const glm::uvec3 grid_dims = FLAMEGPU->environment.getProperty<glm::uvec3>("grid_dims");
        const float N_voxel = static_cast<float>(compMul(grid_dims));
        float total_O2 = FLAMEGPU->environment.getProperty<float>("O2");
        // O2 += (np.sum(Nnbl_grid)+np.sum(Nscl_grid))*envn.P_O20*step_size/(N_voxel*envn.V_grid()*1e-15)/envn.Cs_O2
        const float P_O20 = FLAMEGPU->environment.getProperty<float>("P_O20");
        const unsigned int step_size = FLAMEGPU->environment.getProperty<unsigned int>("step_size");
        const float V_grid = FLAMEGPU->environment.getProperty<float>("V_grid");
        const float Cs_O2 = FLAMEGPU->environment.getProperty<float>("Cs_O2");
        const float P_O2v = FLAMEGPU->environment.getProperty<float>("P_O2v");
        total_O2 += FLAMEGPU->agent("GridCell").sum<unsigned int>("N_l_grid") * P_O20 * step_size / static_cast<float>(N_voxel * V_grid * 1e-15) / Cs_O2;
        total_O2 += P_O2v;
        total_O2 = glm::clamp<float>(total_O2, 0, 1);
        FLAMEGPU->environment.setProperty<float>("O2", total_O2);
    }
}
FLAMEGPU_HOST_FUNCTION(vasculature) {
    // Case 1:
    // Set up the initial vasculature in terms of the amount of oxygen it supplies in one time step.
    // The assumption is that it can supply the amount consumed by the initial population of living neuroblasts and Schwann cells in one time step.
    // Case 2 :
    // When there are more living VEGF - producing neuroblasts than living Schwann cells, an angiogenic signal is produced.
    // When there are enough angiogenic signals, calculate the amount of oxygen consumed by the current population of living cells.
    // Take this as the new oxygen supply rate if it exceeds the old rate.
    const unsigned int t = FLAMEGPU->getStepCounter();  // This should be step_count + 1, however step count is incremented early by calculate_convergence
    const float P_O20 = FLAMEGPU->environment.getProperty<float>("P_O20");
    const unsigned int step_size = FLAMEGPU->environment.getProperty<unsigned int>("step_size");
    const float V_grid = FLAMEGPU->environment.getProperty<float>("V_grid");
    const float Cs_O2 = FLAMEGPU->environment.getProperty<float>("Cs_O2");
    const glm::uvec3 grid_dims = FLAMEGPU->environment.getProperty<glm::uvec3>("grid_dims");
    const unsigned int N_voxel = glm::compMul(grid_dims);
    const int ang_critical = FLAMEGPU->environment.getProperty<int>("ang_critical");
    float P_O2v = FLAMEGPU->environment.getProperty<float>("P_O2v");
    int ang_signal = FLAMEGPU->environment.getProperty<int>("ang_signal");
    unsigned int N_vf = FLAMEGPU->agent("Neuroblastoma").count<int>("VEGF", 1);
    const int NB_COUNT = FLAMEGPU->agent("Neuroblastoma").count();
    const int SC_COUNT = FLAMEGPU->agent("Schwann").count();
    const unsigned int Nnbl_grid = t == 0 ? NB_COUNT : FLAMEGPU->agent("GridCell").sum<unsigned int>("Nnbl_grid");
    const unsigned int Nscl_grid = t == 0 ? SC_COUNT : FLAMEGPU->agent("GridCell").sum<unsigned int>("Nscl_grid");
    if (t == 0) {
        P_O2v = -1.0f * (NB_COUNT + SC_COUNT) * P_O20 * step_size / static_cast<float>(N_voxel * V_grid * 1e-15) / Cs_O2;
        ang_signal = 0;
    } else if (N_vf > Nscl_grid) {
        ang_signal += 1 * step_size;
    }
    if (ang_signal == ang_critical) {
        const float dummy_P_O2v = -1.0f * (Nnbl_grid + Nscl_grid) * P_O20 * step_size / static_cast<float>(N_voxel * V_grid * 1e-15) / Cs_O2;
        if (dummy_P_O2v > P_O2v) {
            P_O2v = dummy_P_O2v;
        }
        ang_signal = 0;
    }
    // Check whether P_O2v is disabled
    const int P_O2v_OFF = FLAMEGPU->environment.getProperty<int>("P_O2v_OFF");
    FLAMEGPU->environment.setProperty<float>("P_O2v", P_O2v_OFF ? 0 : P_O2v);
    FLAMEGPU->environment.setProperty<int>("ang_signal", ang_signal);
}
FLAMEGPU_HOST_FUNCTION(reset_grids) {
    // Reset all grid counters
    // This is mostly useful after cell_cycle, which further manips the grid counters
    FLAMEGPU->environment.getMacroProperty<unsigned int, GMD, GMD, GMD>("Nnbn_grid").zero();
    FLAMEGPU->environment.getMacroProperty<unsigned int, GMD, GMD, GMD>("Nscn_grid").zero();
    FLAMEGPU->environment.getMacroProperty<unsigned int, GMD, GMD, GMD>("Nnbl_grid").zero();
    FLAMEGPU->environment.getMacroProperty<unsigned int, GMD, GMD, GMD>("Nscl_grid").zero();
    FLAMEGPU->environment.getMacroProperty<unsigned int, GMD, GMD, GMD>("Nscl_col_grid").zero();

    // Histograms
    FLAMEGPU->environment.getMacroProperty<unsigned int, GMD, GMD, GMD>("Nnba_grid").zero();
    FLAMEGPU->environment.getMacroProperty<unsigned int, GMD, GMD, GMD>("Nsca_grid").zero();
    FLAMEGPU->environment.getMacroProperty<unsigned int, 42>("histogram_nbl").zero();
    FLAMEGPU->environment.getMacroProperty<unsigned int, 42>("histogram_nba").zero();
    FLAMEGPU->environment.getMacroProperty<unsigned int, 42>("histogram_nbn").zero();
    FLAMEGPU->environment.getMacroProperty<unsigned int, 42>("histogram_scl").zero();
    FLAMEGPU->environment.getMacroProperty<unsigned int, 42>("histogram_sca").zero();
    FLAMEGPU->environment.getMacroProperty<unsigned int, 42>("histogram_scn").zero();

}
FLAMEGPU_HOST_FUNCTION(host_validation) {
    auto GridCell = FLAMEGPU->agent("GridCell");
    auto Neuroblastoma = FLAMEGPU->agent("Neuroblastoma");
    auto Schwann = FLAMEGPU->agent("Schwann");
    const unsigned int validation_Nnbl = FLAMEGPU->environment.getMacroProperty<unsigned int>("validation_Nnbl");
    const unsigned int validation_Nscl = FLAMEGPU->environment.getMacroProperty<unsigned int>("validation_Nscl");
    FLAMEGPU->environment.setProperty<unsigned int>("validation_Nnbl", validation_Nnbl);
    FLAMEGPU->environment.setProperty<unsigned int>("validation_Nscl", validation_Nscl);
    FLAMEGPU->environment.getMacroProperty<unsigned int>("validation_Nnbl").zero();
    FLAMEGPU->environment.getMacroProperty<unsigned int>("validation_Nscl").zero();
    // Cellularity
    const unsigned int NB_living_count = validation_Nnbl;
    const unsigned int NB_apop_count = Neuroblastoma.sum<int>("apop");
    const unsigned int NB_necro_count = Neuroblastoma.sum<int>("necro");
    const unsigned int SC_living_count = validation_Nscl;
    const unsigned int SC_apop_count = Schwann.sum<int>("apop");
    const unsigned int SC_necro_count = Schwann.sum<int>("necro");
    const unsigned int TOTAL_CELL_COUNT = NB_living_count + NB_apop_count + NB_necro_count + SC_living_count + SC_apop_count + SC_necro_count;
    // Calculate each fraction (e.g. number of living SCs/number of all cells) and multiply it by (1-matrix).
    const float ecm = GridCell.sum<float>("matrix_value") / glm::compMul(FLAMEGPU->environment.getProperty<glm::uvec3>("grid_dims"));
    std::array<float, 6> cellularity = {};
    if (TOTAL_CELL_COUNT) {
        cellularity[0] = NB_living_count * (1.0f - ecm) / TOTAL_CELL_COUNT;
        cellularity[1] = NB_apop_count * (1.0f - ecm) / TOTAL_CELL_COUNT;
        cellularity[2] = NB_necro_count * (1.0f - ecm) / TOTAL_CELL_COUNT;
        cellularity[3] = SC_living_count * (1.0f - ecm) / TOTAL_CELL_COUNT;
        cellularity[4] = SC_apop_count * (1.0f - ecm) / TOTAL_CELL_COUNT;
        cellularity[5] = SC_necro_count * (1.0f - ecm) / TOTAL_CELL_COUNT;
    }
    FLAMEGPU->environment.setProperty<float, 6>("validation_cellularity", cellularity);
    // Tumour volume
    float tumour_volume;
    {
        const int total_cell_count = Neuroblastoma.count() + Schwann.count();
        if (total_cell_count) {
            const float rho_tumour = FLAMEGPU->environment.getProperty<float>("rho_tumour");
            const float matrix_dummy = FLAMEGPU->environment.getProperty<float>("matrix_dummy");
            tumour_volume = total_cell_count / rho_tumour / (1 - matrix_dummy);
        } else {
            tumour_volume = FLAMEGPU->environment.getProperty<float>("V_tumour");  // initial tumour volume
        }
        // Convert tumour volume to mm3
        tumour_volume /= 1e+9;
    }
    FLAMEGPU->environment.setProperty<float>("validation_tumour_volume", tumour_volume);
    // Histograms
    std::array<unsigned int, 42> histogram_nbl = {};
    std::array<unsigned int, 42> histogram_nba = {};
    std::array<unsigned int, 42> histogram_nbn = {};
    std::array<unsigned int, 42> histogram_scl = {};
    std::array<unsigned int, 42> histogram_sca = {};
    std::array<unsigned int, 42> histogram_scn = {};

    const auto h_nbl = FLAMEGPU->environment.getMacroProperty<unsigned int, 42>("histogram_nbl");
    const auto h_nba = FLAMEGPU->environment.getMacroProperty<unsigned int, 42>("histogram_nba");
    const auto h_nbn = FLAMEGPU->environment.getMacroProperty<unsigned int, 42>("histogram_nbn");
    const auto h_scl = FLAMEGPU->environment.getMacroProperty<unsigned int, 42>("histogram_scl");
    const auto h_sca = FLAMEGPU->environment.getMacroProperty<unsigned int, 42>("histogram_sca");
    const auto h_scn = FLAMEGPU->environment.getMacroProperty<unsigned int, 42>("histogram_scn");

    for (int i = 0; i < 42; ++i) {
        histogram_nbl[i] = h_nbl[i];
        histogram_nba[i] = h_nba[i];
        histogram_nbn[i] = h_nbn[i];
        histogram_scl[i] = h_scl[i];
        histogram_sca[i] = h_sca[i];
        histogram_scn[i] = h_scn[i];
    }

    FLAMEGPU->environment.setProperty<unsigned int, 42>("histogram_nbl", histogram_nbl);
    FLAMEGPU->environment.setProperty<unsigned int, 42>("histogram_nba", histogram_nba);
    FLAMEGPU->environment.setProperty<unsigned int, 42>("histogram_nbn", histogram_nbn);
    FLAMEGPU->environment.setProperty<unsigned int, 42>("histogram_scl", histogram_scl);
    FLAMEGPU->environment.setProperty<unsigned int, 42>("histogram_sca", histogram_sca);
    FLAMEGPU->environment.setProperty<unsigned int, 42>("histogram_scn", histogram_scn);
}

FLAMEGPU_HOST_FUNCTION(toggle_chemo) {
    int chemo_state = 0;
    int chemo_index = -1;
    const std::array<unsigned int, 336> h_env_chemo_start = FLAMEGPU->environment.getProperty<unsigned int, 336>("chemo_start");
    const std::array<unsigned int, 336> h_env_chemo_end = FLAMEGPU->environment.getProperty<unsigned int, 336>("chemo_end");
    for (unsigned int i = 0; i < sizeof(h_env_chemo_start) / sizeof(h_env_chemo_start[0]); ++i) {
        if (FLAMEGPU->getStepCounter() >= h_env_chemo_start[i]) {
            if (FLAMEGPU->getStepCounter() < h_env_chemo_end[i]) {
                chemo_state = 1;
                chemo_index = i;
                break;
            }
        }
    }
    FLAMEGPU->environment.setProperty<int>("CHEMO_ACTIVE", chemo_state);
    chemo_index *= 6;  // 6 effects per start/end time
    FLAMEGPU->environment.setProperty<int>("CHEMO_OFFSET", chemo_index);
}

FLAMEGPU_HOST_FUNCTION(update_boundary_max) {
    if(FLAMEGPU->getStepCounter()==125){
	const float boundary_max = 1000.0f;
	FLAMEGPU->environment.setProperty<float>("boundary_max", boundary_max);
	const float R_tumour = FLAMEGPU->environment.getProperty<float>("R_tumour");
	FLAMEGPU->environment.setProperty<glm::vec3>("bc_minus", glm::vec3(-boundary_max * R_tumour));
	FLAMEGPU->environment.setProperty<glm::vec3>("bc_plus", glm::vec3(boundary_max * R_tumour));
    }
}
